
#include <hip/hip_runtime.h>
struct UserUpdate
{
	__device__  UserUpdate()
	{
	}
	int update_id;
	int user_id;
	int section;
	float tag;
	float timestamp;
	__device__  UserUpdate(int id, int uid, int sec, float t, float time)
	{
		update_id = id;
		user_id = uid;
		section = sec;
		tag = t;
		timestamp = time;
	}
};

struct FitnessParameter
{
	__device__  FitnessParameter()
	{
	}
	float var1;
	float var2;
	float var3;
	float var4;
	float var5;
	__device__  FitnessParameter(float variable1, float variable2, float variable3, float variable4, float variable5)
	{
		var1 = variable1;
		var2 = variable2;
		var3 = variable3;
		var4 = variable4;
		var5 = variable5;
	}
	__device__  FitnessParameter( float* variables, int variablesLen0)
	{
		var1 = variables[(0)];
		var2 = variables[(1)];
		var3 = variables[(2)];
		var4 = variables[(3)];
		var5 = variables[(4)];
	}
};

struct PredictionPerformances
{
	__device__  PredictionPerformances()
	{
	}
	float occupancyPerformance;
	float occupancyPerformanceRandom;
	float trustPerformance;
	float trustPerformanceRandom;
};

struct SimOptions
{
	__device__  SimOptions()
	{
	}
	float I;
	float lambda_promote;
	float lambda_punish;
	float certainty_coeff;
	float score_coeff;
	float decay;
	__device__  SimOptions(float val_I, float val_lambda_promote, float val_lambda_punish, float val_certainty_coeff, float val_score_coeff, float val_decay)
	{
		I = val_I;
		lambda_promote = val_lambda_promote;
		lambda_punish = val_lambda_punish;
		certainty_coeff = val_certainty_coeff;
		score_coeff = val_score_coeff;
		decay = val_decay;
	}
};


// GeneticAlgorithm.Population
extern "C" __global__  void calculateFitnessOnDevice( float* dev_fitnesses, int dev_fitnessesLen0,  float* groundTruth, int groundTruthLen0, int groundTruthLen1,  float* userTrusts, int userTrustsLen0,  UserUpdate* updates, int updatesLen0, int updatesLen1, int updatesLen2,  FitnessParameter* dev_fitnessParams, int dev_fitnessParamsLen0);
// GeneticAlgorithm.Fitness
__device__  float fitness( float* groundTruth, int groundTruthLen0, int groundTruthLen1,  float* userTrusts, int userTrustsLen0,  UserUpdate* updates, int updatesLen0, int updatesLen1, int updatesLen2, FitnessParameter fitnessParams);
// GeneticAlgorithm.Experiment
__device__  float execute( float* GroundTruth, int GroundTruthLen0, int GroundTruthLen1,  float* UserTrusts, int UserTrustsLen0,  UserUpdate* Updates, int UpdatesLen0, int UpdatesLen1, int UpdatesLen2, FitnessParameter fitnessParam);

// GeneticAlgorithm.Population
extern "C" __global__  void calculateFitnessOnDevice( float* dev_fitnesses, int dev_fitnessesLen0,  float* groundTruth, int groundTruthLen0, int groundTruthLen1,  float* userTrusts, int userTrustsLen0,  UserUpdate* updates, int updatesLen0, int updatesLen1, int updatesLen2,  FitnessParameter* dev_fitnessParams, int dev_fitnessParamsLen0)
{
	int num = blockIdx.x * blockDim.x + threadIdx.x;
	if (num < dev_fitnessesLen0)
	{
		dev_fitnesses[(num)] = fitness(groundTruth, groundTruthLen0, groundTruthLen1, userTrusts, userTrustsLen0, updates, updatesLen0, updatesLen1, updatesLen2, dev_fitnessParams[(num)]);
	}
}
// GeneticAlgorithm.Fitness
__device__  float fitness( float* groundTruth, int groundTruthLen0, int groundTruthLen1,  float* userTrusts, int userTrustsLen0,  UserUpdate* updates, int updatesLen0, int updatesLen1, int updatesLen2, FitnessParameter fitnessParams)
{
	return execute(groundTruth, groundTruthLen0, groundTruthLen1, userTrusts, userTrustsLen0, updates, updatesLen0, updatesLen1, updatesLen2, fitnessParams);
}
// GeneticAlgorithm.Experiment
__device__  float execute( float* GroundTruth, int GroundTruthLen0, int GroundTruthLen1,  float* UserTrusts, int UserTrustsLen0,  UserUpdate* Updates, int UpdatesLen0, int UpdatesLen1, int UpdatesLen2, FitnessParameter fitnessParam)
{
	SimOptions(1.0f, fitnessParam.var1, fitnessParam.var2, fitnessParam.var3, fitnessParam.var4, fitnessParam.var5);
	GroundTruthLen0;
	GroundTruthLen1;
	GroundTruthLen0;
	UpdatesLen1;
	return -1.1f;
}
